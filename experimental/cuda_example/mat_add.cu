#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "glog/logging.h"

#include "experimental/cuda_example/mat_add.h"

namespace qcraft {

static void HandleError(hipError_t err, const char* file, int line) {
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

__global__ void MatAdd(int* a, int* b, int* c, int num) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < num && j < num) {
    int idx = i * num + j;
    c[idx] = a[idx] + b[idx];
  }
}

void MatAddDemo(int num) {
  int m_a[num][num], m_b[num][num], m_c[num][num];
  for (int i = 0; i < num; i++) {
    for (int j = 0; j < num; j++) {
      m_a[i][j] = 1;
      m_b[i][j] = 2;
      m_c[i][j] = 0;
    }
  }

  int total = num * num;
  int *a_gpu, *b_gpu, *c_gpu;

  HANDLE_ERROR(hipMalloc((void**)&a_gpu, total * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&b_gpu, total * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&c_gpu, total * sizeof(int)));

  HANDLE_ERROR(
      hipMemcpy(a_gpu, m_a, total * sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(
      hipMemcpy(b_gpu, m_b, total * sizeof(int), hipMemcpyHostToDevice));

  dim3 grid_size(10, 10);
  dim3 block_size((num + grid_size.x - 1) / grid_size.x,
                  (num + grid_size.y - 1) / grid_size.y);

  MatAdd<<<grid_size, block_size>>>(a_gpu, b_gpu, c_gpu, num);
  HANDLE_ERROR(
      hipMemcpy(m_c, c_gpu, total * sizeof(int), hipMemcpyDeviceToHost));

  // Release device memory
  hipFree(a_gpu);
  hipFree(b_gpu);
  hipFree(c_gpu);
  hipDeviceReset();

  for (int i = 0; i < num; i++) {
    for (int j = 0; j < num; j++) {
      printf("%d ", m_c[i][j]);
    }
    printf("\n");
  }
  printf("\n");
}

}  // namespace qcraft
