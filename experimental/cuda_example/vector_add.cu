#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "glog/logging.h"

#include "experimental/cuda_example/vector_add.h"

namespace qcraft {

static void HandleError(hipError_t err, const char* file, int line) {
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

__global__ void VectorAdd(int* a, int* b, int* c, int num) {
  int i = threadIdx.x;
  if (i < num) {
    c[i] = a[i] + b[i];
  }
}

void VectorAddDemo(int num) {
  int a[num], b[num], c[num];
  for (int i = 0; i < num; i++) {
    a[i] = i;
    b[i] = i * i;
  }

  int block_num = (num + 100 - 1) / 100;
  int thread_num = 100;

  int *a_gpu, *b_gpu, *c_gpu;
  HANDLE_ERROR(hipMalloc((void**)&a_gpu, num * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&b_gpu, num * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&c_gpu, num * sizeof(int)));

  HANDLE_ERROR(hipMemcpy(a_gpu, a, num * sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(b_gpu, b, num * sizeof(int), hipMemcpyHostToDevice));

  VectorAdd<<<block_num, thread_num>>>(a_gpu, b_gpu, c_gpu, num);

  HANDLE_ERROR(hipMemcpy(c, c_gpu, num * sizeof(int), hipMemcpyDeviceToHost));

  for (int i = 0; i < num; i++) {
    printf("%d + %d = %d\n", a[i], b[i], c[i]);
  }

  // Release device memory
  hipFree(a_gpu);
  hipFree(b_gpu);
  hipFree(c_gpu);
  hipDeviceReset();
}

}  // namespace qcraft
